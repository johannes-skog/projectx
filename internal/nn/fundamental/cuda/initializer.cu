#include "hip/hip_runtime.h"
#include <fundamental/tensor.h>
#include <fundamental/cuda/initializer.cuh>

namespace expression{ namespace initilize {

  __global__ void initilize_rand(unsigned int seed, hiprandState_t* states, index_t N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
      hiprand_init(seed, blockIdx.x /* seqeunce idx */, 0 /*offset*/,   &states[idx]);
  }

}}
